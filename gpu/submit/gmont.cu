
#include <hip/hip_runtime.h>
#include <cstring>
#include <string>
#include <iostream>
#include <fstream>
#include <chrono>
#include <iomanip>
#include <sys/time.h>
#include <omp.h>

// 可以自行添加需要的头文件
typedef long long LL;
void fRead(int *a, int *b, int *n, int *p, int input_id){
    // 数据输入函数
    std::string str1 = "./nttdata/";
    std::string str2 = std::to_string(input_id);
    std::string strin = str1 + str2 + ".in";
    char data_path[strin.size() + 1];
    std::copy(strin.begin(), strin.end(), data_path);
    data_path[strin.size()] = '\0';
    std::ifstream fin;
    fin.open(data_path, std::ios::in);
    fin>>*n>>*p;
    for (int i = 0; i < *n; i++){
        fin>>a[i];
    }
    for (int i = 0; i < *n; i++){   
        fin>>b[i];
    }
}

void fCheck(int *ab, int n, int input_id){
    // 判断多项式乘法结果是否正确
    std::string str1 = "./nttdata/";
    std::string str2 = std::to_string(input_id);
    std::string strout = str1 + str2 + ".out";
    char data_path[strout.size() + 1];
    std::copy(strout.begin(), strout.end(), data_path);
    data_path[strout.size()] = '\0';
    std::ifstream fin;
    fin.open(data_path, std::ios::in);
    for (int i = 0; i < n * 2 - 1; i++){
        int x;
        fin>>x;
        if(x != ab[i]){
            std::cout<<"多项式乘法结果错误"<<std::endl;
            return;
        }
    }
    std::cout<<"多项式乘法结果正确"<<std::endl;
    return;
}

void fWrite(int *ab, int n, int input_id){
    // 数据输出函数, 可以用来输出最终结果, 也可用于调试时输出中间数组
    std::string str1 = "files/";
    std::string str2 = std::to_string(input_id);
    std::string strout = str1 + str2 + ".out";
    char output_path[strout.size() + 1];
    std::copy(strout.begin(), strout.end(), output_path);
    output_path[strout.size()] = '\0';
    std::ofstream fout;
    fout.open(output_path, std::ios::out);
    for (int i = 0; i < n * 2 - 1; i++){
        fout<<ab[i]<<'\n';
    }
}

using namespace std;
const int MAXN=300000;
int G=3;
// 类型定义
using u32 = unsigned int;
using i32 = int;
using u64 = unsigned long long;
using i64 = long long;
// 全局变量，模数
u32 m;
// 模的逆元
u32 inv;
// R^2 mod m
u32 R2;

// 计算模的逆元
u32 getinv() {
    u32 inv = m;
    for (int i = 0; i < 4; ++i) {
        inv *= 2 - inv * m;
    }
    return inv;
}

// 蒙哥马利规约核心函数

u32 reduce(u64 x) {
    u32 y = u32(x >> 32) - u32((u64(u32(x) * inv) * m) >> 32);
    return i32(y) < 0? y + m : y;
}
// 将普通整数转换到蒙哥马利域
u32 intToMont(i32 x) {
    return reduce(u64(x) * R2);
}
// 蒙哥马利域内的模加法
u32 Add(u32 x, u32 y) {
    x += y - m;
    return i32(x) < 0? x + m : x;
}

// 蒙哥马利域内的模减法
u32 Dec(u32 x, u32 y) {
    x -= y;
    return i32(x) < 0? x + m : x;
}

// 蒙哥马利域内的模乘法
u32 Mul(u32 x, u32 y) {
    return reduce(u64(x) * y);
}

// 从蒙哥马利域转换回普通整数
i32 get(u32 x) {
    return reduce(x);
}
// 蒙哥马利域内的模幂算法
u32 Pow(u32 base, u32 exponent) {
    u32 result = intToMont(1);
    u32 mont_base = base;

    while (exponent > 0) {
        if (exponent & 1) {
            result = Mul(result, mont_base);
        }
        mont_base = Mul(mont_base, mont_base);
        exponent >>= 1;
    }

    return result;
}
__device__ u32 d_reduce(u64 x,u32 m,u32 inv) {
    u32 y = u32(x >> 32) - u32((u64(u32(x) * inv) * m) >> 32);
    return i32(y) < 0? y + m : y;
}
__device__ u32 d_intToMont(i32 x,u32 R2,u32 m,u32 inv) {
    return d_reduce(u64(x) * R2,m,inv);
}
// 蒙哥马利域内的模加法
__device__ u32 d_Add(u32 x, u32 y,u32 m) {
    x += y - m;
    return i32(x) < 0? x + m : x;
}

// 蒙哥马利域内的模减法
__device__ u32 d_Dec(u32 x, u32 y,u32 m) {
    x -= y;
    return i32(x) < 0? x + m : x;
}

// 蒙哥马利域内的模乘法
__device__ u32 d_Mul(u32 x, u32 y,u32 m,u32 inv) {
    return d_reduce(u64(x) * y,m,inv);
}

// 从蒙哥马利域转换回普通整数
__device__ i32 d_get(u32 x,u32 m,u32 inv) {
    return d_reduce(x,m,inv);
}
__device__ u32 d_Pow(u32 base, u32 exponent,u32 m,u32 inv,u32 R2) {
    u32 result = d_intToMont(1,R2,m,inv);
    u32 mont_base = base;

    while (exponent > 0) {
        if (exponent & 1) {
            result = d_Mul(result, mont_base,m,inv);
        }
        mont_base = d_Mul(mont_base, mont_base,m,inv);
        exponent >>= 1;
    }

    return result;
}

// 位逆序置换核函数（基础版）
__global__ void bit_reversal_kernel(u32 *data, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;
    
    // 确定n的有效位数m（即n-1的最高位位置+1）
    int m = 0;
    int nn = n - 1;
    while (nn > 0) {
        nn >>= 1;
        m++;
    }
    
    // 计算逆序后的索引
    int reversed = 0;
    int temp = idx;
    for (int i = 0; i < m; i++) {
        reversed = (reversed << 1) | (temp & 1);
        temp >>= 1;
    }
    
    // 交换元素（只处理idx < reversed的情况，避免重复交换）
    if (idx < reversed && reversed < n) {
        u32 temp = data[idx];
        data[idx] = data[reversed];
        data[reversed] = temp;
    }
}
// 点乘运算核函数
__global__ void multiply_kernel(u32 *a, u32 *b, u32 *c, int n, int p,u32 m,u32 inv) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = d_Mul(a[idx] , b[idx],m,inv); // 模运算确保结果在域内
    }
}
// 点乘运算核函数
__global__ void get_kernel(u32 *a,int n,u32 m,u32 inv) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        a[idx] = d_get(a[idx],m,inv); // 模运算确保结果在域内
    }
}
__global__ void intToMont_kernel(u32 *fa, int *a, u32 *fb, int *b, int n, u32 R2,u32 m,u32 inv) {
        
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if (idx < n) 
{
// 并行转换a和b数组到Montgomery域
fa[idx] = d_intToMont(a[idx], R2,m,inv);
fb[idx] = d_intToMont(b[idx], R2,m,inv);
}
}
// 元素乘以常数的核函数
__global__ void multiply_const_kernel(u32 *a, int constant, int n, int p,u32 m,u32 inv) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        a[idx] = d_Mul(a[idx] , constant,m,inv);
    }
}
__global__ void roots_kernel(int g, int p, int k, int current_len, u32* h_wn, u32* h_wn_inv,u32 m,u32 inv,u32 R2) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (j < current_len / 2) {
        int root = d_Pow(g, (p - 1) / current_len,m,inv,R2);
        int root_inv = d_Pow(root, p - 2, m,inv,R2);

        int w1 = d_Pow(root, j, m,inv,R2);
        int w2 = d_Pow(root_inv, j, m,inv,R2);

        h_wn[j * (k / current_len)] = w1;
        h_wn_inv[j * (k / current_len)] = w2;
    }
}
// 蝶形运算核函数 - 全局内存版本
__global__ void ntt_butterfly_kernel(u32 *a, u32 *wn, int n, int p, int len,u32 m,u32 inv) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n/2) return;
    
    int half_len = len / 2;
    int i = (idx / half_len) * len;
    int j = idx % half_len;
    
    // 确保索引不会越界
    if (i + j + half_len >= n) return;
    
    int twiddle_idx = j * (n / len);
    int u = a[i + j];
    int v =d_Mul(wn[twiddle_idx] , a[i + j + half_len],m,inv);
    
    a[i + j] = d_Add(u , v,m);
    a[i + j + half_len] = d_Dec(u , v ,m) ;
}

void ntt(u32 *d_a, int n, int p, int inv_flag,u32 *d_wn,u32 m,u32 inv) {
    
    u32 g=intToMont(G);
    // 位逆序置换
    dim3 block_size(512);
    dim3 grid_size((n + block_size.x - 1) / block_size.x);
    bit_reversal_kernel<<<grid_size, block_size>>>(d_a, n);
    //cudaDeviceSynchronize();

    // 蝶形运算
    for (int len = 2; len <= n; len <<= 1) {
        ntt_butterfly_kernel<<<grid_size, block_size>>>(d_a, d_wn, n, p, len,m,inv);
        //cudaDeviceSynchronize();
    }
 
     
    if (inv_flag) {
        // 计算 n 关于 p 的逆元 inv_n
        u32 inv_n = Pow(intToMont(n), p- 2);
        u32 mont_inv_n = inv_n;
        multiply_const_kernel<<<grid_size, block_size>>>(d_a, mont_inv_n, n, p,m,inv);
        hipDeviceSynchronize();
    }
}

// 多项式乘法函数，利用 NTT 实现
void poly_multiply(int *a, int *b, int *ab, int n, int p) {
    //m=intToMont(p);
    m=p;
    inv = getinv();
    // 计算 R^2 mod m
    R2 = -u64(m) % m;
   
    int k = 1;
    while (k < 2 * n) {
        k <<= 1;
    }

    dim3 block_size(512);
    dim3 grid_size((k + block_size.x - 1) / block_size.x);
    u32 *d_a,*d_b, *d_wn,*d_wn_inv,*d_result;
    u32 *h_wn = new u32[k];
    u32 *h_wn_inv=new u32[k];
    int* fa,*fb;
    // 分配GPU内存
    hipMalloc((void**)&d_a, k * sizeof(u32));
    hipMalloc((void**)&d_b, k * sizeof(u32));
    hipMalloc((void**)&d_wn, k * sizeof(u32));
    hipMalloc((void**)&d_wn_inv, k * sizeof(u32));
    hipMalloc((void**)&d_result, k * sizeof(u32));
    hipMalloc((void**)&fa, k * sizeof(int));
    hipMalloc((void**)&fb, k * sizeof(int));
     // 使用主机端快速幂计算root
     u32 g=intToMont(G);
     for (int len = 2; len <= k; len <<= 1) {
        int num_j_values = len / 2;
        int blockSize = 256;
        int gridSize = (num_j_values + blockSize - 1) / blockSize;
        roots_kernel<<<gridSize, blockSize>>>(g, p, k, len, d_wn, d_wn_inv,m,inv,R2);

    }
   
    // 复制数据到GPU
    hipMemcpy(fa, a, k * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(fb, b, k * sizeof(int), hipMemcpyHostToDevice);
    intToMont_kernel<<<grid_size, block_size>>>(d_a, fa, d_b, fb, k, R2,m,inv);
    ntt(d_a, k, p, false,d_wn,m,inv);
    ntt(d_b, k, p, false,d_wn,m,inv);
    // 点乘运算
    multiply_kernel<<<grid_size, block_size>>>(d_a, d_b, d_result, k, p,m,inv);
    hipDeviceSynchronize();
    
    // 逆向变换
    ntt(d_result, k, p, true,d_wn_inv,m,inv);

    
    get_kernel<<<grid_size, block_size>>>(d_result,k,m,inv);
    // 复制结果回CPU
    hipMemcpy(ab, d_result, k * sizeof(u32), hipMemcpyDeviceToHost);
    // 释放GPU内存
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_wn);
    hipFree(d_result);
}
int a[300000], b[300000], ab[300000];
int main(int argc, char *argv[])
{
    
    // 保证输入的所有模数的原根均为 3, 且模数都能表示为 a \times 4 ^ k + 1 的形式
    // 输入模数分别为 7340033 104857601 469762049 263882790666241
    // 第四个模数超过了整型表示范围, 如果实现此模数意义下的多项式乘法需要修改框架
    // 对第四个模数的输入数据不做必要要求, 如果要自行探索大模数 NTT, 请在完成前三个模数的基础代码及优化后实现大模数 NTT
    // 输入文件共五个, 第一个输入文件 n = 4, 其余四个文件分别对应四个模数, n = 131072
    // 在实现快速数论变化前, 后四个测试样例运行时间较久, 推荐调试正确性时只使用输入文件 1
    int test_begin = 0;
    int test_end = 3;
    for(int i = test_begin; i <= test_end; ++i){
        long double ans = 0;
        int n_, p_;
        memset(a,0,sizeof(a));
        memset(b,0,sizeof(b));
        memset(ab,0,sizeof(ab));
        fRead(a, b, &n_, &p_, i);
        auto Start = std::chrono::high_resolution_clock::now();
        // TODO : 将 poly_multiply 函数替换成你写的 ntt
        poly_multiply(a, b, ab, n_, p_);
        auto End = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double,std::ratio<1,1000>>elapsed = End - Start;
        ans += elapsed.count();
        fCheck(ab, n_, i);
        std::cout<<"average latency for n = "<<n_<<" p = "<<p_<<" : "<<ans<<" (us) "<<std::endl;
        // 可以使用 fWrite 函数将 ab 的输出结果打印到 files 文件夹下
        // 禁止使用 cout 一次性输出大量文件内容
        fWrite(ab, n_, i);
    }
    return 0;
}
